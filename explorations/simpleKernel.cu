
#include <hip/hip_runtime.h>
extern "C"
__global__ void 
getIndex(int *out, int N)
{
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) {
      out[idx] = idx;
    }
}
